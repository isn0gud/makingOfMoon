#include "hip/hip_runtime.h"
#include "RndTestSimGPU.cuh"
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "../../util/hip/hip_runtime_api.h"

__global__ void update(glm::vec4 *pPos) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < NUM_PARTICLES) {
        pPos[i] = pPos[i] + glm::vec4(0.01 * pPos[i].x, 0.01 * pPos[i].y,
                                      0.01 * pPos[i].z, 1);
//        printf("test\n");
    }
}


void RndTestSimGPU::updateStep(int numTimeSteps) {

    size_t size = particles->numParticles * sizeof(glm::vec4);
    glm::vec4 *d_particles;

    checkCudaErrors(hipGraphicsMapResources(1, &vboParticlesPos_cuda));

    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **) &d_particles,
                                                   &size, vboParticlesPos_cuda));


    // Update the position of the particles
    update <<< 256, 256 >>> (d_particles);

    checkCudaErrors(hipDeviceSynchronize());

    // Unmap the SSBO to be available to OpenGL
    checkCudaErrors(hipGraphicsUnmapResources(1, &vboParticlesPos_cuda));
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

RndTestSimGPU::RndTestSimGPU(Particles *particles, hipGraphicsResource_t particlePos)
        : particles(particles),
          vboParticlesPos_cuda(particlePos) {}

