#include "hip/hip_runtime.h"
#include "RndTestSimGPU.cuh"
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <>

__global__ void update(glm::vec4 *pPos) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 3000) {
        pPos[i] = pPos[i] + glm::vec4(0.01 * pPos[i].x, 0.01 * pPos[i].y,
                                      0.01 * pPos[i].z, 1);
//        printf("test\n");
    }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
///src: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


void RndTestSimGPU::updateStep(int numTimeSteps) {

    size_t size = particles->numParticles * sizeof(glm::vec4);
    glm::vec4 *d_particles;

    gpuErrchk(hipGraphicsMapResources(1, &vboParticlesPos_cuda));

    gpuErrchk(hipGraphicsResourceGetMappedPointer((void **) &d_particles,
                                                   &size, vboParticlesPos_cuda));


    // Update the position of the particles
    update <<< 256, 256 >>> (d_particles);


    // Unmap the SSBO to be available to OpenGL
    gpuErrchk(hipGraphicsUnmapResources(1, &vboParticlesPos_cuda));
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

RndTestSimGPU::RndTestSimGPU(Particles *particles, hipGraphicsResource_t particlePos) : particles(particles),
                                                                                         vboParticlesPos_cuda(particlePos) {

}

